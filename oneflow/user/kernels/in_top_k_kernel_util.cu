#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/user/kernels/in_top_k_kernel_util.h"

namespace oneflow {

namespace {

// template<typename T, typename K>

template<typename T, typename K>
__global__ void InTopkGpu(const int targets_num, const int classes_num, const T* targets,
                          const K* predictions, const int k, int8_t* out) {
  CUDA_1D_KERNEL_LOOP(batch_idx, targets_num) {
    T target = targets[batch_idx];
    bool cannot_say =
        (target >= classes_num) || !isfinite(predictions[batch_idx * classes_num + target]);

    int32_t more_probable_classes = 0;
    if (!cannot_say) {
      const K target_prediction = predictions[batch_idx * classes_num + target];
      FOR_RANGE(int32_t, class_idx, 0, classes_num) {
        K pred = predictions[batch_idx * classes_num + class_idx];

        if (!isfinite(pred)) {
          cannot_say = true;
          break;
        } else if (pred > target_prediction) {
          ++more_probable_classes;
          if (more_probable_classes > k) break;
        }
      }
    }
    out[batch_idx] = cannot_say ? false : (more_probable_classes < k);
  }
}

}  // namespace

template<typename T, typename K>
struct InTopkKernelUtil<DeviceType::kGPU, T, K> {
  static void InTopk(DeviceCtx* ctx, const int targets_num, const int classes_num, const T* targets,
                     const K* predictions, const int k, int8_t* out) {
    RUN_CUDA_KERNEL((InTopkGpu<T, K>), ctx, targets_num, targets_num, classes_num, targets,
                    predictions, k, out);
  }
};

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_IN_TOP_K_FUNCTOR, (DeviceType::kGPU),
                                 INDEX_DATA_TYPE_SEQ, OF_PP_MAKE_TUPLE_SEQ(float, DataType::kFloat))

}  // namespace oneflow
